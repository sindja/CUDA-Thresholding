#include "hip/hip_runtime.h"
#include <iostream>
#include "lodepng.h"
#include "lodepng.c"
#include "hip/hip_runtime.h"
#include ""


__global__ void color_balancing(unsigned char * input,int* min, int* max)
{
	int id = blockIdx.x * blockDim.x + threadIdx.x;
	for (int i = 0; i < 3; ++i)
	{
		float temp = 255 / (float)(max[i] - min[i]);
		input[id * 3 + i] = (input[id * 3 + i]<min[i]) ? min[i] : (input[id * 3 + i]>max[i]) ? max[i] : input[id * 3 + i];
		input[id * 3 + i] = (int)((input[id * 3 + i] - min[i]) *temp);
	}


}
__global__ void hist_equalisation(unsigned char * input, int* eq_hist1, int* eq_hist2, int* eq_hist3)
{
	int id = blockIdx.x * blockDim.x + threadIdx.x;
	
		input[id * 3 ] = eq_hist1[input[id * 3]];
		input[id * 3+1] = eq_hist2[input[id * 3+1]];
		input[id * 3+2] = eq_hist3[input[id * 3+2]];

}
__global__ void kernelapplyFilter(unsigned char * image, unsigned char * output_image, float* filter, int filterDim, int imageInH, int imageInW)
{	
	float sumR = 0, sumG = 0, sumB = 0;
	int idT = blockIdx.x * blockDim.x + threadIdx.x;
	if (idT > imageInW && idT < (imageInH - 1)*imageInW)
	{
		int i = idT / (imageInW);
		int j = idT % (imageInW);
		int k = filterDim / 2;
		
		for (int fi = -k; fi <= k; fi++)
		{
			for (int fj = -k; fj <= k; fj++)
			{
				sumR += filter[(fi + k)*filterDim + fj + k] * image[((i + fi)*(imageInW)+j + fj) * 3];
				sumG += filter[(fi + k)*filterDim + fj + k] * image[((i + fi)*(imageInW)+j + fj) * 3 + 1];
				sumB += filter[(fi + k)*filterDim + fj + k] * image[((i + fi)*(imageInW)+j + fj) * 3 + 2];
			}
		}

		output_image[(i*(imageInW)+j) * 3] = (int) ((sumR > 255) ? 255 : sumR);
		output_image[(i*(imageInW)+j) * 3 + 1] = (int) ((sumG > 255) ? 255 : sumG);
		output_image[(i*(imageInW)+j) * 3 + 2] = (int) ((sumB > 255) ? 255 : sumB);
	}
}
__global__ void kernelNegative(unsigned char * image)
{
	int id = blockIdx.x * blockDim.x + threadIdx.x;
	image[id * 3] = ~image[id * 3];
	image[id * 3 + 1] = ~image[id * 3 + 1];
	image[id * 3 + 2] = ~image[id * 3 + 2];
}
__global__ void kernelLogartithmic(unsigned char * image, double constant)
{
	int id = blockIdx.x * blockDim.x + threadIdx.x;
	int idB = id * 3;
	int r = constant * (__logf(image[idB]) + 1);
	int g = constant * (__logf(image[idB + 1]) + 1);
	int b = constant * (__logf(image[idB + 2]) + 1);
	image[idB] = (r>255) ? 255 : r;
	image[idB + 1] = (g>255) ? 255 : g;
	image[idB + 2] = (b>255) ? 255 : b;
}
__global__ void kernelPowLow(unsigned char * image, double constant, double gamma)
{
	int id = blockIdx.x * blockDim.x + threadIdx.x;
	int idB = id * 3;
	int r = constant * (powf(image[idB], gamma));
	int g = constant * (powf(image[idB + 1], gamma));
	int b = constant * (powf(image[idB + 2], gamma));
	image[idB] = (r > 255) ? 255 : r;
	image[idB + 1] = (g > 255) ? 255 : g;
	image[idB + 2] = (b > 255) ? 255 : b;
}
__global__ void kernelPiecewise(unsigned char* image, int down, int up)
{
	int id = blockIdx.x * blockDim.x + threadIdx.x;
	int idB = id * 3;
	int sum = (image[idB] * 0.3 + image[idB + 1] * 0.59 + image[idB + 2] * 0.11)/2.55;
	if (sum < down || sum > up)
	{
		image[idB] = 80;
		image[idB + 1] = 80;
		image[idB + 2] = 80;
	}
	else
	{
		image[idB] = 160;
		image[idB + 1] = 160;
		image[idB + 2] = 160;
	}

}


int main(int argc, char ** argv)
{


	int blockSize = 256;
	int gridSize;
	unsigned char * array;
	int *min, *max;
	size_t pngsize;
	 unsigned char *png;
	float *d_filter;
	const char * filename = "Lenna_test.png";
	lodepng_load_file(&png, &pngsize, filename);

	unsigned char *image,*output_image;
	unsigned int width, height;
	//ucitavanje slike
	unsigned int error = lodepng_decode24(&image, &width, &height, png, pngsize);


	if (error != 0){
		std::cout << "error " << error << ": " << lodepng_error_text(error) << std::endl;
	}
	//ukupan broj piksela u slici
	unsigned int N = width*height;

	//histogrami: pocetni, kumulativni, ujednaceni
	 int hist[3][255];
	 int cumul_hist[3][255];
	 int equal_hist[3][255];

	for (int i = 0; i < 255; i++)
	{
		hist[0][i] = 0;
		cumul_hist[0][i] = 0;
		hist[1][i] = 0;
		cumul_hist[1][i] = 0;
		hist[2][i] = 0;
		cumul_hist[2][i] = 0;
	}
	//formiranje sva tri histograma
	for (int i = 0; i < N; i++)
	{
		hist[0][image[3 * i]]++;
		hist[1][image[3 * i+1]]++;
		hist[2][image[3 * i+2]]++;
	}

	cumul_hist[0][0] = hist[0][0];
	cumul_hist[1][0] = hist[1][0];
	cumul_hist[2][0] = hist[2][0];
	for (int i = 0; i < 3; i++)
	{
		for (int j = 1; j < 255; j++)
		{
			cumul_hist[i][j] = cumul_hist[i][j - 1] + hist[i][j];
		}
	}
	float temp[3];
	 temp[0] = 255.0 / cumul_hist[0][254];
	 temp[1] = 255.0 / cumul_hist[1][254];
	 temp[2] = 255.0 / cumul_hist[2][254];
	 for (int i = 0; i < 3; i++)
	 {
		 for (int j = 0; j < 255; j++)
		 {
			 equal_hist[i][j] = cumul_hist[i][j] * temp[i]+0.5;
		 }
	 }
	 
	 int operation = 0;
	 printf(" Filteri:\n");
	 printf(" 1: Balansiranje boje \n 2: Ujednacavanje histograma \n 3: Konvolucija \n 4: Negativ \n 5: Logaritamska transformacija \n 6: Gama transformacija \n 7: Naglasavanje dela \n 8: Blurovanje slike \n");
	 printf("---------------------------\n");
	 printf(" Unesite broj zeljenog filtera: ");

	 scanf("%d", &operation);
	
	 gridSize = N / blockSize;
	 switch (operation)
	 {
		 //Color balancing
		 case 1:
		 {
				   int s1, s2;
				   printf("\n Unesite procente za odsecanje: \n");
				   //printf(" Donja granica: ");
				   scanf("%d ", &s1);
				   //printf("\n");
				   //printf(" Gornja granica: ");
				   //fflush(stdin);
				   scanf("%d", &s2);
				   //printf("\n");

				   int vmin[3] = { 0, 0, 0 };
				   int vmax[3] = { 254, 254, 254 };
				   float temp_min = N* ((float)s1 / 100);
				   float  temp_max = N*((float)1 - (float)s2 / 100);
				   for (int i = 0; i < 3; i++)
				   {
					   while (cumul_hist[i][vmin[i] + 1] <= temp_min)
						   vmin[i]++;
					   while (cumul_hist[i][vmax[i] - 1] > temp_max)
						   vmax[i]--;
					   if (vmax[i] < 255 - 1)
						   vmax[i]++;
				   }
				   hipMalloc((void **)& array, sizeof (char)* width*height * 3);
				   hipMemcpy(array, image, sizeof (char)* width*height * 3, hipMemcpyHostToDevice);
				   hipMalloc((void **)& max, sizeof (int)* 3);
				   hipMemcpy(max, vmax, sizeof (int)* 3, hipMemcpyHostToDevice);
				   hipMalloc((void **)& min, sizeof (int)* 3);
				   hipMemcpy(min, vmin, sizeof (int)* 3, hipMemcpyHostToDevice);
				   color_balancing << <gridSize, blockSize >> > (array, min, max);
				   hipMemcpy(image, array, sizeof (char)* width*height * 3, hipMemcpyDeviceToHost);

				   hipFree(array);
				   hipFree(min);
				   hipFree(max);
		 }
			 break;
		 //Ujednacavanje histograma
		 case 2:
		 {
				   int * eq_hist1, *eq_hist2, *eq_hist3;
				   hipMalloc((void **)& array, sizeof (char)* width*height * 3);
				   hipMemcpy(array, image, sizeof (char)* width*height * 3, hipMemcpyHostToDevice);
				
				   hipMalloc((void **)& eq_hist1, sizeof (int)* 255*3);
				   hipMemcpy(eq_hist1, equal_hist[0], sizeof (int)*255, hipMemcpyHostToDevice);
				   
				   hipMalloc((void **)& eq_hist2, sizeof (int)* 255 * 3);
				   hipMemcpy(eq_hist2, equal_hist[1], sizeof (int)* 255, hipMemcpyHostToDevice);
				   hipMalloc((void **)& eq_hist3, sizeof (int)* 255 * 3);
				   hipMemcpy(eq_hist3, equal_hist[2], sizeof (int)* 255, hipMemcpyHostToDevice);
				   hist_equalisation << <gridSize, blockSize >> > (array, eq_hist1, eq_hist2, eq_hist3);
				   hipMemcpy(image, array, sizeof (char)* width*height * 3, hipMemcpyDeviceToHost);

				   hipFree(array);
				   hipFree(eq_hist1);
				   hipFree(eq_hist2);
				   hipFree(eq_hist3);
		 }
			 break;
		//Konvolucija
		 case 3:
		 {		
				   int filter_num;
				   printf(" Konvolucioni kerneli:\n");
				   printf(" 1: Naglasavanje ivica \n 2: Naglasavanje ivica 2 \n 3: Laplasov kernel \n 4: Izostravanje \n 5: Emboss \n 6: Prosecna vrednost \n");
				   printf(" Unesite broj zeljenog konvolucionog kernela: ");
				   scanf("%d", &filter_num);
				   float filter1[9] = { 0, 1, 0, 1, -4, 1, 0, 1, 0 };
				   float filter2[9] = { 1, 0, -1, 0, 0, 0, -1, 0, 1 };
				   float filter3[9] = { -1, -1, -1, -1, 8, -1, -1, -1, -1 }; // laplacian filter
				   float filter4[9] = { 0, -1, 0, -1, 5, -1, 0, -1, 0 };
				   float filter5[9] = { -2, -1, 0, -1, 1, 1, 0, 1, 2 };//emboss
				   float filter6[9] = { 1.0f / 9.0f, 1.0f / 9.0f, 1.0f / 9.0f, 1.0f / 9.0f, 1.0f / 9.0f, 1.0f / 9.0f, 1.0f / 9.0f, 1.0f / 9.0f, 1.0f / 9.0f };//low pass filter
				   float* filter;
				   switch (filter_num)
				   {
					   case 1:
					   {
								 filter = filter1;
					   }
						   break;
					   case 2:
					   {
								 filter = filter2;
					   }
						   break;
					   case 3:
					   {
								 filter = filter3;
					   }
						   break;
					   case 4:
					   {
								 filter = filter4;
					   }
						   break;
					   case 5:
					   {
								 filter = filter5;
					   }
						   break;
					   case 6:
					   {
								 filter = filter6;
					   }
						   break;

					   default:
							filter = filter1;
						   break;
				   }

				   int filterDim = 3;
				   
				   hipMalloc((void **)& array, sizeof (char)* height*width* 3);
				   hipMemcpy(array, image, sizeof (char)* height*width * 3, hipMemcpyHostToDevice);
				   hipMalloc((void **)& output_image, sizeof (char)*width*height * 3);
				   
				 
				   hipMalloc((void **)&d_filter, sizeof (float)* 9);
				   hipMemcpy(d_filter, filter, sizeof (float)* 9, hipMemcpyHostToDevice);
				  
				   kernelapplyFilter << <gridSize, blockSize >> > (array, output_image, d_filter, filterDim,height,width);
				   hipMemcpy(image, output_image, sizeof (char)* width*height * 3, hipMemcpyDeviceToHost);

				   hipFree(array);
				   hipFree(output_image);
				   hipFree(d_filter);

		 }
			 break;
		//Negativ
		 case 4:
		 {
				   hipMalloc((void **)& array, sizeof (char)* height*width * 3);
				   hipMemcpy(array, image, sizeof (char)* height*width * 3, hipMemcpyHostToDevice);
				   kernelNegative << <gridSize, blockSize >> >(array);
				   hipMemcpy(image, array, sizeof (char)* width*height * 3, hipMemcpyDeviceToHost);
				   hipFree(array);
		 } 
			 break;
		//Logaritamska transformacija
		 case 5:
			 {		
				   double constant = 0;
				   printf("%s", "Unesite zeljenu vrednost konstante za logaritamsku transformaciju: ");
				   scanf("%lf", &constant);
				   //printf("%f\n", constant);
				   hipMalloc((void **)& array, sizeof (char)* height*width * 3);
				   hipMemcpy(array, image, sizeof (char)* height*width * 3, hipMemcpyHostToDevice);
				   kernelLogartithmic << <gridSize, blockSize >> >(array, constant);
				   hipMemcpy(image, array, sizeof (char)* width*height * 3, hipMemcpyDeviceToHost);
				   hipFree(array);
			 }
			 break;
		//Gama transformacija
		 case 6:
		 {
				   double constant = 0, gama = 0;;
				   printf("%s\n", "Unesite zeljenu vrednost konstante za ovu transformaciju: ");
				   scanf("%lf", &constant);
				   printf("%s\n", "Unesite zeljenu vrednost gama za ovu transformaciju: ");
				   scanf("%lf", &gama);
				   hipMalloc((void **)& array, sizeof (char)* height*width * 3);
				   hipMemcpy(array, image, sizeof (char)* height*width * 3, hipMemcpyHostToDevice);
				   kernelPowLow << <gridSize, blockSize >> >(array, constant,gama);
				   hipMemcpy(image, array, sizeof (char)* width*height * 3, hipMemcpyDeviceToHost);
				   hipFree(array);
		 }
			 break;
		//Naglasavanje dela
		 case 7:
		 {
				   
					   int down = 0, up = 0;;
					   printf("%s", "Unesite zeljenu vrednost donje granice osvetljenosti u procentima: ");
					   scanf("%d", &down);
					   printf("%s", "Unesite zeljenu vrednost gornje granice osvetljenosti u procentima: ");
					   scanf("%d", &up);
					   hipMalloc((void **)& array, sizeof (char)* height*width * 3);
					   hipMemcpy(array, image, sizeof (char)* height*width * 3, hipMemcpyHostToDevice);
					   kernelPiecewise << <gridSize, blockSize >> >(array, down, up);
					   hipMemcpy(image, array, sizeof (char)* width*height * 3, hipMemcpyDeviceToHost);
					   hipFree(array);
				 
		 }
			 break;
		// Blurovanje slike
		 case 8:
		 {
				   int count;
				   printf("%s", "Unesite koliko puta zelite da primenite blur efekat na sliku: ");
				   scanf("%d", &count);

				   float filter[9] = { 1.0f / 9.0f, 1.0f / 9.0f, 1.0f / 9.0f, 1.0f / 9.0f, 1.0f / 9.0f, 1.0f / 9.0f, 1.0f / 9.0f, 1.0f / 9.0f, 1.0f / 9.0f };//low pass filter
				   int filterDim = 3;

				   hipMalloc((void **)& array, sizeof (char)* height*width * 3);
				   hipMalloc((void **)& output_image, sizeof (char)*width*height * 3);
				   hipMalloc((void **)&d_filter, sizeof (float)* 9);

				   for (int i = 0; i < count; i++)
				   {
					   hipMemcpy(array, image, sizeof (char)* height*width * 3, hipMemcpyHostToDevice);
					   hipMemcpy(d_filter, filter, sizeof (float)* 9, hipMemcpyHostToDevice);
					   kernelapplyFilter << <gridSize, blockSize >> > (array, output_image, d_filter, filterDim, height, width);
					   hipMemcpy(image, output_image, sizeof (char)* width*height * 3, hipMemcpyDeviceToHost);
				   }

				   hipFree(array);
				   hipFree(output_image);
				   hipFree(d_filter);

		 }
			 break;
	 }
	 hipError_t errSync = hipGetLastError();
	 hipError_t errAsync = hipDeviceSynchronize();
	 if (errSync != hipSuccess)
		 printf("Sync kernel error: %s\n", hipGetErrorString(errSync));
	 if (errAsync != hipSuccess)
		 printf("Async kernel error: %s\n", hipGetErrorString(errAsync));
	 hipFree(array);
	lodepng_encode24_file("lenna_processed.png", image, width, height);

	return 0;
} 

